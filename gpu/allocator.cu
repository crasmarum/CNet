#include "gpu_data.h"
#include "allocator.h"

long GpuHelper::getTotalGlobalMem() {
	int nDevices;	
	hipGetDeviceCount(&nDevices);
	
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		return prop.totalGlobalMem;
	}
	
	return 0;
}

/**
 * Assume one device for now.
 */
int GpuHelper::getMaxThreadsPerBlock() {
	int nDevices;	
	hipGetDeviceCount(&nDevices);
	
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		return prop.maxThreadsPerBlock;
	}
	
	return 0;
}



